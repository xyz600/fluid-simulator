#include "hip/hip_runtime.h"
// based on https://gist.github.com/kamino410/09df4ecdf37b03cbd05752a7b2e52d3a
// this adds ImGui to an application with CUDA and OpenGL. the thing is, once you use CUDA, ImGui renders very strangely.
// after 9 hours of debugging I found that putting glBindBuffer before and after the draw call fixes this!
// glBindBuffer(GL_PIXEL_UNPACK_BUFFER, pbo); // THE MAGIC LINE #1
// glDrawPixels(WIDTH, HEIGHT, GL_RGBA, GL_UNSIGNED_BYTE, 0);
// glBindBuffer(GL_PIXEL_UNPACK_BUFFER, 0);   // THE MAGIC LINE #2

#include "imgui.h"
#include "imgui_impl_glfw.h"
#include "imgui_impl_opengl3.h"

#include <GL/glew.h> // GL
#include <GLFW/glfw3.h>
#include <cuda_gl_interop.h>
#include <hip/hip_runtime.h> // CUDA
#include <>
#include <iostream>
#include <stdlib.h>

#define WIDTH 512 * 2
#define HEIGHT 512

GLuint pbo;
hipGraphicsResource* cudapbo;

#define gpuErrchk(ans)                        \
    {                                         \
        gpuAssert((ans), __FILE__, __LINE__); \
    }
inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = true)
{
    if (code != hipSuccess) {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort)
            exit(code);
    }
}

__global__ void kernel(uchar4* map, unsigned char frame)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int id = x + y * blockDim.x * gridDim.x;

    map[id].x = x / 2;
    map[id].y = y / 2;
    map[id].z = frame;
    map[id].w = 255;
}

extern "C" void kernelUpdate(int width, int height)
{
    static unsigned char frame = 0;
    frame++;
    uchar4* dev_map;

    gpuErrchk(hipGraphicsMapResources(1, &cudapbo, NULL));
    gpuErrchk(hipGraphicsResourceGetMappedPointer((void**)&dev_map, NULL, cudapbo));

    dim3 threads(8, 8);
    dim3 grids(width / 8, height / 8);
    kernel<<<grids, threads>>>(dev_map, frame);

    gpuErrchk(hipDeviceSynchronize());
    gpuErrchk(hipGraphicsUnmapResources(1, &cudapbo, NULL));
}

int main()
{
    // GLFW + OpenGL
    if (!glfwInit())
        exit(EXIT_FAILURE);

    if (atexit(glfwTerminate)) {
        glfwTerminate();
        exit(EXIT_FAILURE);
    }
    const char* glsl_version = "#version 130";
    glfwWindowHint(GLFW_CONTEXT_VERSION_MAJOR, 3);
    glfwWindowHint(GLFW_CONTEXT_VERSION_MINOR, 0);
    GLFWwindow* window = glfwCreateWindow(WIDTH, HEIGHT, "gl-cuda-test", NULL, NULL);
    if (!window)
        exit(EXIT_FAILURE);

    glfwMakeContextCurrent(window);
    glfwSwapInterval(1);
    

    if (glewInit() != GLEW_OK)
        exit(EXIT_FAILURE);

    // ImGui
    IMGUI_CHECKVERSION();
    ImGui::CreateContext();
    ImGui::StyleColorsDark(); // ImGui::StyleColorsClassic();
    ImGui_ImplGlfw_InitForOpenGL(window, true);
    ImGui_ImplOpenGL3_Init(glsl_version);
    bool show_demo_window = true;
    bool show_another_window = false;
    ImVec4 clear_color = ImVec4(0.45f, 0.55f, 0.60f, 1.00f);

    // CUDA with GL interop
    glGenBuffers(1, &pbo); // make & register PBO
    glBindBuffer(GL_PIXEL_UNPACK_BUFFER, pbo);
    glBufferData(GL_PIXEL_UNPACK_BUFFER, 4 * sizeof(GLubyte) * WIDTH * HEIGHT, NULL, GL_DYNAMIC_DRAW);
    gpuErrchk(hipGraphicsGLRegisterBuffer(&cudapbo, pbo, hipGraphicsRegisterFlagsWriteDiscard));

    while (!glfwWindowShouldClose(window)) {

        kernelUpdate(WIDTH, HEIGHT);
        glBindBuffer(GL_PIXEL_UNPACK_BUFFER, pbo); // THE MAGIC LINE #1
        glDrawPixels(WIDTH, HEIGHT, GL_RGBA, GL_UNSIGNED_BYTE, 0);
        glBindBuffer(GL_PIXEL_UNPACK_BUFFER, 0); // THE MAGIC LINE #2

        glfwPollEvents();

        // Start the Dear ImGui frame
        ImGui_ImplOpenGL3_NewFrame();
        ImGui_ImplGlfw_NewFrame();
        ImGui::NewFrame();

        // 1. Show the big demo window (Most of the sample code is in ImGui::ShowDemoWindow()! You can browse its code to learn more about Dear ImGui!).
        if (show_demo_window)
            ImGui::ShowDemoWindow(&show_demo_window);

        // 2. Show a simple window that we create ourselves. We use a Begin/End pair to created a named window.
        {
            static float f = 0.0f;
            static int counter = 0;
            ImGui::Begin("Hello, world!"); // Create a window called "Hello, world!" and append into it.
            ImGui::Text("This is some useful text."); // Display some text (you can use a format strings too)
            ImGui::Checkbox("Demo Window", &show_demo_window); // Edit bools storing our window open/close state
            ImGui::Checkbox("Another Window", &show_another_window);
            ImGui::SliderFloat("float", &f, 0.0f, 1.0f); // Edit 1 float using a slider from 0.0f to 1.0f
            ImGui::ColorEdit3("clear color", (float*)&clear_color); // Edit 3 floats representing a color
            if (ImGui::Button("Button")) // Buttons return true when clicked (most widgets return true when edited/activated)
                counter++;
            ImGui::SameLine();
            ImGui::Text("counter = %d", counter);
            ImGui::Text("Application average %.3f ms/frame (%.1f FPS)", 1000.0f / ImGui::GetIO().Framerate, ImGui::GetIO().Framerate);
            ImGui::End();
        }

        // 3. Show another simple window.
        {
            ImGui::Begin("Another Window", &show_another_window); // Pass a pointer to our bool variable (the window will have a closing button that will clear the bool when clicked)
            ImGui::Text("Hello from another window!");
            if (ImGui::Button("Close Me"))
                show_another_window = false;
            ImGui::End();
        }

        // Rendering
        ImGui::Render();
        ImGui_ImplOpenGL3_RenderDrawData(ImGui::GetDrawData());
        glfwSwapBuffers(window);
    }

    glBindBuffer(GL_PIXEL_UNPACK_BUFFER, 0); // exit
    gpuErrchk(cudaGLUnregisterBufferObject(pbo));
    gpuErrchk(hipGraphicsUnregisterResource(cudapbo));
    glDeleteBuffers(1, &pbo);
    glfwTerminate();
    return 0;
}
